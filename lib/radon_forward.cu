#include "hip/hip_runtime.h"
#include <cassert>
#include <cstdint>

#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

static constexpr int warp_size = 32;
static constexpr int n_threads = 256;
static constexpr int n_warps = n_threads / warp_size;

__device__ __inline__
static float y_rotated(float theta, float x, float y)
{
	return sin(theta) * x + cos(theta) * y;
}

__device__ __inline__
static void vrange(float *vmin, float *vmax, float theta, float x0, float x1, float y0, float y1)
{
	for(; theta < 0; theta += 2.0 * M_PI){}
	for(; theta >= 2.0 * M_PI; theta -= 2.0 * M_PI){}

	if(theta < 0.5 * M_PI){
		*vmin = y_rotated(theta, x0, y0);
		*vmax = y_rotated(theta, x1, y1);
	}else if(theta < M_PI){
		*vmin = y_rotated(theta, x0, y1);
		*vmax = y_rotated(theta, x1, y0);
	}else if(theta < 1.5 * M_PI){
		*vmin = y_rotated(theta, x1, y1);
		*vmax = y_rotated(theta, x0, y0);
	}else{
		*vmin = y_rotated(theta, x1, y0);
		*vmax = y_rotated(theta, x0, y1);
	}
}

__device__ __inline__
static void projection(float *acc,
		hipTextureObject_t texObj, int width, int height,
		float xc, float yc, float theta,
		float u, float v){
	float x_ = xc + cos(theta) * u + sin(theta) * v;
	float y_ = yc - sin(theta) * u + cos(theta) * v;

	if(x_ >= 0 && x_ < (float)width){
		if(y_ >= 0 && y_ < (float)height){
			*acc += tex2D<float>(texObj, x_, y_);
		}
	}
}

__global__
static void radonT_gpu_calc(float *sino, hipTextureObject_t tomo,
		int width, int height, int umax,
		int n_angles, float *angles,
		float xc, float yc, float uc)
		
{
	int u = blockDim.x * blockIdx.x + threadIdx.x;
	int t = blockIdx.y;
	float theta = angles[t];
	float v_min, v_max;
	vrange(&v_min, &v_max, theta, -xc, (float)(width-1)-xc, -yc, (float)(height-1)-yc);

	float acc = 0.0;
	for(float v = v_min; v < v_max; v += 1.0){
		projection(&acc, tomo, width, height, xc, yc, theta, (float)u - uc, v);
	}

	if(u < umax){
		sino[umax * t + u] = acc;
	}
}

void radonT_gpu(float *sino, const float *tomo,
		int width, int height, int umax,
		int n_angles, float *angles,
		float xc, float yc, float uc)
{
	hipChannelFormatDesc channelDesc =
		hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
	hipArray *tomo_;
	hipMallocArray(&tomo_, &channelDesc, width, height);
	hipMemcpy2DToArray(tomo_, 0, 0, tomo, width*sizeof(float), width*sizeof(float), height, hipMemcpyDeviceToDevice);

	struct hipResourceDesc resDesc;
	memset(&resDesc, 0, sizeof(resDesc));
	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = tomo_;

	struct hipTextureDesc texDesc;
	memset(&texDesc, 0, sizeof(texDesc));
	texDesc.addressMode[0] = hipAddressModeWrap;
	texDesc.addressMode[1] = hipAddressModeWrap;
	texDesc.filterMode = hipFilterModeLinear;
	texDesc.readMode = hipReadModeElementType;
	texDesc.normalizedCoords = false;

	hipTextureObject_t texObj = 0;
	hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL);

	dim3 block(n_threads);
	dim3 grid((width + block.x - 1) / block.x, n_angles, 1);
	radonT_gpu_calc<<<grid, block>>>(sino, texObj, width, height, umax, n_angles, angles, xc, yc, uc);

	hipDestroyTextureObject(texObj);
	hipFreeArray(tomo_);
}

torch::Tensor radon_cuda_forward(
		torch::Tensor tomo,
		torch::Tensor angles,
		int width_sino,
		float x_center,
		float y_center,
		float u_center)
{
	AT_DISPATCH_FLOATING_TYPES(tomo.type(), "radon_cuda_forward", ([&] {
		if(sizeof(scalar_t) != 32){
			AT_ERROR("radon_cuda_forward is implemented for only 32-bit floating point");
		}else{
			int height = tomo.size(0);
			int width = tomo.size(1);
			int n_angles = angles.size(0);

			auto options = torch::TensorOptions()
				.dtype(torch::kFloat32)
				.device(torch::kCUDA, tomo.device().index());
			torch::Tensor sino = torch::empty({n_angles, width_sino}, options);

			radonT_gpu(
				sino.data_ptr<float>(),
				tomo.data_ptr<float>(),
				width,
				height,
				width_sino,
				n_angles,
				angles.data_ptr<float>(),
				x_center,
				y_center,
				u_center);
			return sino;
		}
	}));
}
